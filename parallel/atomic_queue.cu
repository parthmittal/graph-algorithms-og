#include "hip/hip_runtime.h"
#include "atomic_queue.cuh"

__device__ void atomic_push(queue_t Q, int element)
{
	int idx = atomicAdd(Q.head, 1);
	Q.data[idx] = element;
}
