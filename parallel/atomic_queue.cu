#include "hip/hip_runtime.h"
__device__ void atomic_push(queue_t Q, int element)
{
	int idx = atomicAdd(Q.head, 1);
	Q.data[idx] = element;
}
